#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>


#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define ALIVE 1
#define DEAD 0

#  define CUDA_SAFE_CALL( call) {                                    \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } }

int termCheck(int *prev, int *next, int N, int M, int gen);
__global__ void kernel_update(int* t,int* t1,int N,int M);
__global__ void initdat(int *t, int *t1, int N, int M, time_t clock);

int main(int argc, char *argv[]) {

	int N,				/* rows of the grid */
		M,				/* columns of the grid */
		Gens,			/* amount of generations. */
		perGens;		/* checks termination per perGens generations. if perGens zero no termination check */
	int *grid, *grid1;
	int *gpu_grid, *gpu_grid1, *gpu_temp;

	if ( argc != 5) {
		printf("Error! Missing mandatory argument.\n");
		return 1;
	}

	N = atoi(argv[1]); /* Getting rows amount */
	M = atoi(argv[2]); /* Getting columns amount */
	Gens = atoi(argv[3]); /* Getting Gens */
	perGens = atoi(argv[4]);
	if (Gens <= 0 || N < 0 || M < 0 || perGens < 0) {
		printf("Please give positive values for rows/cols and Generations\n");
		return 1;
	}

	int blockSize = 512;
	int numBlocks = (N*M + blockSize - 1) / blockSize;

	grid = (int*)malloc(sizeof(int)*N*M);
	grid1 = (int*)malloc(sizeof(int)*N*M);

	CUDA_SAFE_CALL(hipMalloc(&gpu_grid, N*M*sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc(&gpu_grid1, N*M*sizeof(int)));

	/* Initialize random data */
	initdat<<<numBlocks,blockSize>>>(gpu_grid, gpu_grid1, N, M, time(NULL));
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	for (int k = 1; k <= Gens; k++) {

		kernel_update<<<numBlocks,blockSize>>>(gpu_grid,gpu_grid1,N,M);
		CUDA_SAFE_CALL(hipDeviceSynchronize());

		if ( perGens ) {
			CUDA_SAFE_CALL(hipMemcpy(grid, gpu_grid, N*M*sizeof(int), hipMemcpyDeviceToHost));
			CUDA_SAFE_CALL(hipMemcpy(grid1, gpu_grid1, N*M*sizeof(int), hipMemcpyDeviceToHost));

			if ( k % perGens == 0) {
				if (termCheck(grid, grid1, N, M, k)) {
					hipFree(gpu_grid1);
					hipFree(gpu_grid);
					free(grid);
					free(grid1);
					return 0;
				}
			}
		}
		gpu_temp = gpu_grid;
		gpu_grid = gpu_grid1;
		gpu_grid1 = gpu_temp;
	}
	printf("Reached requested generations %d\n",Gens );

	hipFree(gpu_grid1);
	hipFree(gpu_grid);
	free(grid);
	free(grid1);

	return 0;
}

int termCheck(int *prev, int *next, int N, int M, int gen){
	int allDiff = 0;
	int sum = 0;
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < M; j++) {
			if (prev[i*M+j] != next[i*M+j]) {
				allDiff = 1;
			}
			sum += next[i*M+j];
		}
	}
	if (!sum) {
		printf("All cells are dead at generation %d\n", gen);
		return 1;
	}
	else if (!allDiff) {
		printf("Generation %d is the same with generation %d\n", gen, gen-1);
		return 1;
	}
	return 0;
}


__global__ void kernel_update(int* t,int* t1,int N,int M){

    int x = blockIdx.x * blockDim.x + threadIdx.x;

    /*update starts*/
    if (0 <= x && x < N*M ){     //if not out of bounds then..
        int i,j,neighbours;
        i = x / M;
        j = x % M;
        if (i+1 > N-1) {
            if (j-1 < 0) {
                /* eimai o bot_left */
				neighbours = t[(i-1)*M+M-1] + t[(i-1)*M+j] + t[(i-1)*M+j+1] + t[i*M+j+1] + t[0*M+j+1] + t[0*M+j] + t[0*M+M-1] + t[i*M+M-1];
			}
            else if (j+1 > M-1) {
                /* eimai o bot_right */
				neighbours = t[(i-1)*M+j-1] + t[(i-1)*M+j] + t[(i-1)*M+0] + t[i*M+0] + t[0*M+0] + t[0*M+j] + t[0*M+j-1] + t[i*M+j-1];
			}
            else{
                /* eimai aplos bot */
				neighbours = t[(i-1)*M+j-1] + t[(i-1)*M+j] + t[(i-1)*M+j+1] + t[i*M+j+1] + t[0*M+j+1] + t[0*M+j] + t[0*M+j-1] + t[i*M+j-1];
			}
        }
        else if (i-1 < 0) {
            if (j-1 < 0) {
                /* eimai o top_left */
				neighbours = t[(N-1)*M+M-1] + t[(N-1)*M+j] + t[(N-1)*M+j+1] + t[i*M+j+1] + t[(i+1)*M+j+1] + t[(i+1)*M+j] + t[(i+1)*M+M-1] + t[i*M+M-1];
			}
            else if (j+1 > M-1) {
                /* eimai o top_right */
				neighbours = t[(N-1)*M+j-1] + t[(N-1)*M+j] + t[(N-1)*M+0] + t[i*M+0] + t[(i+1)*M+0] + t[(i+1)*M+j] + t[(i+1)*M+j-1] + t[i*M+j-1];
			}
            else{
                /* eimai aplos top */
				neighbours = t[(N-1)*M+j-1] + t[(N-1)*M+j] + t[(N-1)*M+j+1] + t[i*M+j+1] + t[(i+1)*M+j+1] + t[(i+1)*M+j] + t[(i+1)*M+j-1] + t[i*M+j-1];
			}
        }
        else if (j-1 < 0) {
            /* eimai aplos left */
			neighbours = t[(i-1)*M+M-1] + t[(i-1)*M+j] + t[(i-1)*M+j+1] + t[i*M+j+1] + t[(i+1)*M+j+1] + t[(i+1)*M+j] + t[(i+1)*M+M-1] + t[i*M+M-1];
		}
        else if (j+1 > M-1) {
            /* eimai aplos right */
			neighbours = t[(i-1)*M+j-1] + t[(i-1)*M+j] + t[(i-1)*M+0] + t[i*M+0] + t[(i+1)*M+0] + t[(i+1)*M+j] + t[(i+1)*M+j-1] + t[i*M+j-1];
		}
        else{
            /* oi geitones mou den peftoun eksw */
			neighbours = t[(i-1)*M+j-1] + t[(i-1)*M+j] + t[(i-1)*M+j+1] + t[i*M+j+1] + t[(i+1)*M+j+1] + t[(i+1)*M+j] + t[(i+1)*M+j-1] + t[i*M+j-1];
		}
        /* kanones paixnidiou edw */
        if (t[x] == ALIVE) {
            if (neighbours <= 1 || neighbours >= 4) {
                t1[x] = DEAD;
            }
            else{
                t1[x] = ALIVE;
            }
        }
        else if (t[x] == DEAD && neighbours == 3) {
            t1[x] = ALIVE;
        }
		else{
			t1[x] = DEAD;
		}
    }
}

__global__ void initdat(int *t, int *t1, int N, int M, time_t clock){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState_t state;
	hiprand_init(clock,x,0,&state);


	if (0 <= x && x < N*M ){
		t[x] = (hiprand(&state) % 4) ? DEAD : ALIVE;
		t1[x] = DEAD;
	}

}
